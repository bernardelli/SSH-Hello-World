#include "little_cuda_functions.h"

void checkingDevices()
{
	int deviceCount;
	hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
	
	if (cudaStatus == hipErrorInsufficientDriver) {
		fprintf(stderr, "hipGetDeviceCount failed!  Do you have CUDA installed?\n");
		exit(1);
	}
	
	int device;
	for (device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		
		if(device<100) printf("Device %d with name %s has compute capability %d.%d and concurrentKernels = %d.\n",
			device, deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.concurrentKernels);
	}
	hipDeviceReset();

	

	
}

hipError_t allocateGpuMemory(float**ptr, int size)
{
	hipError_t cudaStatus = hipMalloc((float**)ptr, size * sizeof(float));
	return cudaStatus;
}

hipError_t copyToGpuMem(float *a, float *b, int size)
{
	hipError_t cudaStatus = hipMemcpy(a, b, size * sizeof(float), hipMemcpyHostToDevice);
	return cudaStatus;
}
